#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/csrc/distributed/c10d/Utils.hpp>
#include <torch/torch.h>
#include <algorithm>

namespace c10d {

// CUDA kernel to check if data has NAN, device side assert
// is raised if NAN is found
template <typename T>
__global__ void checkForNaN(T* data, size_t size) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;

  for (size_t i = tid; i < size; i += stride) {
    CUDA_KERNEL_ASSERT(!isnan(data[i]));
  }
}

// CHECK if a Tensor contains NAN in any of its element
void checkForNan(const at::Tensor& tensor) {
  // skip check for non float types
  if (!torch::is_floating_point(tensor)) {
    return;
  }
  const size_t maxNumThreadsPerBlock = 256;
  const size_t maxNumBlocks = 24;
  const size_t numThreadsPerBlock =
      std::min<size_t>(maxNumThreadsPerBlock, tensor.numel());

  const size_t numBlocks = std::min<size_t>(
      maxNumBlocks,
      (tensor.numel() + numThreadsPerBlock - 1) / numThreadsPerBlock);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(tensor.scalar_type(), "checkForNaN", [&] {
    checkForNaN<scalar_t><<<numBlocks, numThreadsPerBlock>>>(
        tensor.data_ptr<scalar_t>(), tensor.numel());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });

}

} // namespace c10d
